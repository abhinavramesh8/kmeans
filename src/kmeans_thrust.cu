#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/fill.h>
#include <thrust/adjacent_difference.h>
#include <thrust/binary_search.h>
#include <thrust/find.h>
#include <thrust/reduce.h>
#include <thrust/tuple.h>

#include <chrono>

#include "helpers.h"

struct unary_modulus {
    int a;
    
    __host__ __device__
        int operator()(const int& x) const {
            return x % a;
        }
};

struct unary_divide {
    int a;
    
    __host__ __device__
        int operator()(const int& x) const {
            return x / a;
        }
};

struct binary_divide {
    __host__ __device__
        float operator()(const float& x, const int& y) const {
            if (y != 0) {
                return x / y;
            }
            return x;
        }
};

struct point_element_index {
    int dims;
    int n_cent_eles;
    
    __host__ __device__
        int operator()(const int& i) const {
            return (dims * (i / n_cent_eles) + i % dims);
        }
};

struct squared_difference {
    __host__ __device__
        float operator()(const thrust::tuple<float, float>& x) const {
            float diff = thrust::get<0>(x) - thrust::get<1>(x);
            return diff * diff;
        }
};

struct arg_min {
    __host__ __device__
        thrust::tuple<int, float> operator()(const thrust::tuple<int, float>& x, 
                                             const thrust::tuple<int, float>& y) const {
            if (thrust::get<1>(x) < thrust::get<1>(y)) {
                return x;
            }
            else {
                return y;
            }
        }
};

struct first_element {
    __host__ __device__
        int operator()(const thrust::tuple<int, float>& x) const {
            return thrust::get<0>(x);
        }
};

struct centroid_element_index {
    int dims;
    
    __host__ __device__
        int operator()(const thrust::tuple<int, int>& x) const {
            return (thrust::get<0>(x) + dims * thrust::get<1>(x));
        }
};

struct square_root {
    __host__ __device__
        float operator()(const float& x) const {
            return sqrtf(x);
        }
};

struct greater_than {
    float a;
    
    __host__ __device__
        bool operator()(const float& x) const {
            return (x > a);
        }
};

static void assign_centroids(const thrust::device_vector<float>& pt_eles,
                             const thrust::device_vector<float>& cent_eles,
                             thrust::device_vector<int>& labels, int n_points,
                             int n_clusters, int dims) {
    static int n_cent_eles = n_clusters * dims;
    static int n_pt_cent_pairs = n_points * n_clusters;
    static int n_pt_cent_eles = n_pt_cent_pairs * dims;
    static thrust::counting_iterator<int> counter_beg(0);
    static auto pt_ele_idx_beg = thrust::make_transform_iterator(counter_beg, 
                                                                 point_element_index{dims, 
                                                                                     n_cent_eles});
    static auto pt_ele_beg = thrust::make_permutation_iterator(pt_eles.begin(), 
                                                               pt_ele_idx_beg);
    static auto cent_ele_idx_beg = thrust::make_transform_iterator(counter_beg, 
                                                                   unary_modulus{n_cent_eles});
    static auto cent_ele_beg = thrust::make_permutation_iterator(cent_eles.begin(), 
                                                                 cent_ele_idx_beg);
    static auto pt_cent_ele_beg = thrust::make_zip_iterator(thrust::make_tuple(pt_ele_beg, 
                                                                               cent_ele_beg));
    static auto sq_diff_beg = thrust::make_transform_iterator(pt_cent_ele_beg, 
                                                              squared_difference{});
    static auto idx_beg = thrust::make_transform_iterator(counter_beg, unary_divide{dims});
    thrust::device_vector<float> sq_dists(n_pt_cent_pairs);
    thrust::device_vector<int> output_keys(n_pt_cent_pairs);
    thrust::reduce_by_key(idx_beg, idx_beg + n_pt_cent_eles, sq_diff_beg, 
                          output_keys.begin(), sq_dists.begin());
    output_keys.resize(n_points);
    static auto pt_idx_beg = thrust::make_transform_iterator(counter_beg, 
                                                             unary_divide{n_clusters});
    static auto cent_idx_beg = thrust::make_transform_iterator(counter_beg, 
                                                               unary_modulus{n_clusters});
    auto cent_idx_sq_dist_beg = thrust::make_zip_iterator(thrust::make_tuple(
        cent_idx_beg, sq_dists.begin()));
    thrust::device_vector<thrust::tuple<int, float> > labels_sq_dists(n_points);
    thrust::reduce_by_key(pt_idx_beg, pt_idx_beg + n_pt_cent_pairs, cent_idx_sq_dist_beg,
                          output_keys.begin(), labels_sq_dists.begin(), 
                          thrust::equal_to<int>(), arg_min{});
    thrust::transform(labels_sq_dists.begin(), labels_sq_dists.end(), 
                      labels.begin(), first_element{});
}

static void recompute_centroids(thrust::device_vector<float>& cent_eles,
                                thrust::device_vector<int>& labels,
                                const thrust::device_vector<float>& pt_eles,
                                int n_clusters, int n_points, int dims) {
    static thrust::counting_iterator<int> counter_beg(0);
    static auto idx_beg = thrust::make_transform_iterator(counter_beg, 
                                                          unary_divide{dims});
    static auto cent_idx_beg = thrust::make_permutation_iterator(labels.begin(), 
                                                                 idx_beg);
    static auto dim_beg = thrust::make_transform_iterator(counter_beg, 
                                                          unary_modulus{dims});
    static auto dim_cent_idx_beg = thrust::make_zip_iterator(thrust::make_tuple(
        dim_beg, cent_idx_beg));
    static auto cent_ele_idx_beg = thrust::make_transform_iterator(dim_cent_idx_beg, 
                                                                   centroid_element_index{dims});
    thrust::device_vector<float> dev_pt_eles = pt_eles;
    static int n_pt_eles = n_points * dims;
    thrust::device_vector<int> cent_ele_idx(cent_ele_idx_beg, cent_ele_idx_beg + n_pt_eles);
    thrust::sort_by_key(cent_ele_idx.begin(), cent_ele_idx.end(), dev_pt_eles.begin());
    static int n_cent_eles = n_clusters * dims;
    thrust::device_vector<int> output_keys(n_cent_eles);
    thrust::reduce_by_key(cent_ele_idx.begin(), cent_ele_idx.end(), dev_pt_eles.begin(),
                          output_keys.begin(), cent_eles.begin());
    thrust::device_vector<int> counts(n_clusters);
    thrust::sort(labels.begin(), labels.end());
    thrust::upper_bound(labels.begin(), labels.end(), counter_beg, 
                        counter_beg + n_clusters, counts.begin());
    thrust::adjacent_difference(counts.begin(), counts.end(), counts.begin());
    auto cent_count_beg = thrust::make_permutation_iterator(counts.begin(), idx_beg);
    thrust::transform(cent_eles.begin(), cent_eles.end(), cent_count_beg, 
                      cent_eles.begin(), binary_divide{});
}

static bool converged(const thrust::device_vector<float>& cent_eles, 
                      thrust::device_vector<float>& old_cent_eles, float threshold, 
                      int n_clusters, int dims) {
    static int n_cent_eles = n_clusters * dims;
    static auto cent_old_cent_ele_beg = thrust::make_zip_iterator(thrust::make_tuple(
        cent_eles.begin(), old_cent_eles.begin()));
    static auto sq_diff_beg = thrust::make_transform_iterator(cent_old_cent_ele_beg, 
                                                              squared_difference{});
    static thrust::counting_iterator<int> counter_beg(0);
    static auto cent_idx_beg = thrust::make_transform_iterator(counter_beg, 
                                                               unary_divide{dims});
    thrust::device_vector<float> distances(n_clusters);
    thrust::device_vector<int> cent_idx(n_clusters);
    thrust::reduce_by_key(cent_idx_beg, cent_idx_beg + n_cent_eles, 
                          sq_diff_beg, cent_idx.begin(), 
                          distances.begin());
    thrust::transform(distances.begin(), distances.end(), distances.begin(), 
                      square_root{});
    auto iter = thrust::find_if(distances.begin(), distances.end(), 
                                greater_than{threshold});
    return (iter == distances.end());
}

void kmeans_thrust(kmeans_args_t& args) {
    // auto all_start = std::chrono::high_resolution_clock::now();
    thrust::device_vector<float> pt_eles(args.point_elements, 
                                         args.point_elements + args.n_points * args.dims);
    int n_cent_eles = args.n_clusters * args.dims;
    // auto init_start = std::chrono::high_resolution_clock::now();
    thrust::device_vector<float> cent_eles(args.centroid_elements, 
                                           args.centroid_elements + n_cent_eles);
    thrust::device_vector<float> old_cent_eles(n_cent_eles);
    thrust::device_vector<int> labels(args.n_points);
    // auto init_end = std::chrono::high_resolution_clock::now();
    bool done = false;
    int iters = 0;
    std::chrono::milliseconds iter_time(0);
    while (!done) {
        auto start = std::chrono::high_resolution_clock::now();
        thrust::copy(cent_eles.begin(), cent_eles.end(), old_cent_eles.begin());
        iters++;
        assign_centroids(pt_eles, cent_eles, labels, args.n_points, 
                         args.n_clusters, args.dims);
        recompute_centroids(cent_eles, labels, pt_eles, args.n_clusters, 
                            args.n_points, args.dims);
        done = (iters >= args.max_iters) || converged(cent_eles, old_cent_eles, 
                                                      args.threshold, 
                                                      args.n_clusters, args.dims);
        auto end = std::chrono::high_resolution_clock::now();
        auto diff = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
        iter_time += diff;
    }
    // auto fin_start = std::chrono::high_resolution_clock::now();
    thrust::copy(cent_eles.begin(), cent_eles.end(), args.centroid_elements);
    thrust::copy(pt_eles.begin(), pt_eles.end(), args.point_elements);
    thrust::copy(labels.begin(), labels.end(), args.labels);
    // auto fin_end = std::chrono::high_resolution_clock::now();
    args.iters = iters;
    args.iter_time = static_cast<float>(iter_time.count());
    args.iter_time /= args.iters;
//     auto all_end = std::chrono::high_resolution_clock::now();
//     auto diff = std::chrono::duration_cast<std::chrono::milliseconds>(all_end - all_start);
//     cout << "End-to-end: " << diff.count() << " ms\n";
//     diff = std::chrono::duration_cast<std::chrono::milliseconds>(init_end - init_start);
//     cout << "Initial data transfer: " << diff.count() << " ms\n";
//     diff = std::chrono::duration_cast<std::chrono::milliseconds>(fin_end - fin_start);
//     cout << "Final data transfer: " << diff.count() << " ms\n";
}