#include "hip/hip_runtime.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

#include "helpers.h"
#include "kmeans_cuda_basic.h"

__device__ __host__ float distance(float *x, float *y, int dims) {
    float sq_dist = 0.0f;
    for (int i = 0; i < dims; i++) {
        float diff = x[i] - y[i];
        sq_dist += (diff * diff);
    }
    return sqrtf(sq_dist);
}

__device__ void add_vector(float *to_vect, float *from_vect, int dims) {
    for (int i = 0; i < dims; i++) {
        atomicAdd(to_vect + i, from_vect[i]);
    }
}

__global__ void assign_centroid(float *pt_eles, float *cent_eles, int *labels, 
                                int n_points, int n_clusters, int dims) {
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx < n_points) {
        float *point = pt_eles + pt_idx * dims;
        float min_dist = HIP_INF_F;
        int min_idx;
        for (int i = 0; i < n_clusters; i++) {
            float dist = distance(point, cent_eles + dims * i, dims);
            if (dist < min_dist) {
                min_dist = dist;
                min_idx = i;
            }
        }
        labels[pt_idx] = min_idx;
    }
}

__global__ void zero_centroid_element(float *cent_eles, int n_cent_eles) {
    int cent_ele_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (cent_ele_idx < n_cent_eles) {
        cent_eles[cent_ele_idx] = 0.0f;
    }
}

__global__ void add_point_to_centroid(float *cent_eles, float *pt_eles, int *labels, 
                                      int *counts, int n_points, int dims) {
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx < n_points) {
        int cent_idx = labels[pt_idx];
        add_vector(cent_eles + dims * cent_idx, pt_eles + pt_idx * dims, dims);
        atomicAdd(counts + cent_idx, 1);
    }
}

__global__ void average_centroid_element(float *cent_eles, int *counts, 
                                         int n_cent_eles, int dims) {
    int cent_ele_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (cent_ele_idx < n_cent_eles) {
        int count = counts[cent_ele_idx / dims];
        if (count != 0) {
            cent_eles[cent_ele_idx] /= count;
        }
    }
}

__global__ void centroid_converged(float *cent_eles, float *old_cent_eles, 
                                   bool *converged, float threshold, 
                                   int n_clusters, int dims) {
    int cent_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (cent_idx < n_clusters) {
        int offset = dims * cent_idx;
        float dist = distance(cent_eles + offset, old_cent_eles + offset, dims);
        if (dist > threshold) {
            converged[cent_idx] = false;
        } else {
            converged[cent_idx] = true;
        }
    }
}

void kmeans_cuda_basic(kmeans_args_t& args) {
    float iter_diff = 0.0f;
    hipEvent_t iter_start, iter_stop;
//     float all_diff = 0.0f, init_diff = 0.0f, end_diff = 0.0f, tran1_diff = 0.0f, 
//           tran2_diff = 0.0f;
//     hipEvent_t all_start, all_stop, init_start, init_stop, end_start, end_stop, 
//                 tran1_start, tran1_stop, tran2_start, tran2_stop;
//     hipEventCreate(&all_start);
//     hipEventCreate(&all_stop);
//     hipEventCreate(&init_start);
//     hipEventCreate(&init_stop);
//     hipEventCreate(&end_start);
//     hipEventCreate(&end_stop);
//     hipEventCreate(&tran1_start);
//     hipEventCreate(&tran1_stop);
//     hipEventCreate(&tran2_start);
//     hipEventCreate(&tran2_stop);
    hipEventCreate(&iter_start);
    hipEventCreate(&iter_stop);
//     hipEventRecord(all_start);
//     hipEventRecord(init_start);
    
    int n_cent_eles = args.n_clusters * args.dims;
    int cent_eles_sz = n_cent_eles * sizeof(float);
    float *dev_centroid_elements;
    hipMalloc((void**)&dev_centroid_elements, cent_eles_sz);
    hipMemcpy(dev_centroid_elements, args.centroid_elements, cent_eles_sz,
               hipMemcpyHostToDevice);
    
    float *old_centroid_elements;
    hipMalloc((void**)&old_centroid_elements, cent_eles_sz);
     
    int pt_eles_sz = args.n_points * args.dims * sizeof(float);
    float *dev_point_elements;
    hipMalloc((void**)&dev_point_elements, pt_eles_sz);
    hipMemcpy(dev_point_elements, args.point_elements, pt_eles_sz,
               hipMemcpyHostToDevice);
      
    int *dev_counts;
    int counts_sz = args.n_clusters * sizeof(int);
    hipMalloc((void**)&dev_counts, counts_sz);
    
    int *dev_labels;
    int labels_sz = args.n_points * sizeof(int);
    hipMalloc((void**)&dev_labels, labels_sz);
    
    int conv_sz = args.n_clusters * sizeof(bool);
    bool *converged = (bool*) malloc(conv_sz);
    bool *dev_converged;
    hipMalloc((void**)&dev_converged, conv_sz);
    
//     hipEventRecord(init_stop);
//     hipEventSynchronize(init_stop);
//     hipEventElapsedTime(&init_diff, init_start, init_stop);
    
    int iters = 0;
    bool done = false;
    while (!done) {
        hipEventRecord(iter_start);
//         hipEventRecord(tran1_start);
        hipMemcpy(old_centroid_elements, dev_centroid_elements, cent_eles_sz,
                   hipMemcpyDeviceToDevice);
//         hipEventRecord(tran1_stop);
//         hipEventSynchronize(tran1_stop);
        float diff;
//         hipEventElapsedTime(&diff, tran1_start, tran1_stop);
//         tran1_diff += diff;
        
        iters++;
        
        hipMemset(dev_counts, 0, counts_sz);
        
        int n_threads_per_block = 256;
        int n_blocks_pts = (int)ceil(args.n_points / (float)n_threads_per_block);
        int n_blocks_cent_eles = (int)ceil(n_cent_eles / (float)n_threads_per_block);
        int n_blocks_cents = (int)ceil(args.n_clusters / (float)n_threads_per_block);
        
        assign_centroid<<<n_blocks_pts, n_threads_per_block>>>(
            dev_point_elements, dev_centroid_elements, dev_labels, args.n_points, 
            args.n_clusters, args.dims);
        
        zero_centroid_element<<<n_blocks_cent_eles, n_threads_per_block>>>(
            dev_centroid_elements, n_cent_eles);
        
        add_point_to_centroid<<<n_blocks_pts, n_threads_per_block>>>(
            dev_centroid_elements, dev_point_elements, dev_labels, dev_counts, 
            args.n_points, args.dims);
        
        average_centroid_element<<<n_blocks_cent_eles, n_threads_per_block>>>(
            dev_centroid_elements, dev_counts, n_cent_eles, args.dims);
        
        centroid_converged<<<n_blocks_cents, n_threads_per_block>>>(
            dev_centroid_elements, old_centroid_elements, dev_converged, 
            args.threshold, args.n_clusters, args.dims);
        
//         hipEventRecord(tran2_start);
        hipMemcpy(converged, dev_converged, conv_sz, hipMemcpyDeviceToHost);
//         hipEventRecord(tran2_stop);
//         hipEventSynchronize(tran2_stop);
//         hipEventElapsedTime(&diff, tran2_start, tran2_stop);
//         tran2_diff += diff;
        hipEventRecord(iter_stop);
        hipEventSynchronize(iter_stop);
        hipEventElapsedTime(&diff, iter_start, iter_stop);
        iter_diff += diff;
        bool centroids_converged = true;
        for (int i = 0; i < args.n_clusters; i++) {
            if (!converged[i]) {
                centroids_converged = false;
                break;
            }
        }
        done = (iters >= args.max_iters) || centroids_converged;
    }
    args.iters = iters;
    args.iter_time = iter_diff;
    args.iter_time /= iters;
    
//     hipEventRecord(end_start);
    hipMemcpy(args.point_elements, dev_point_elements, pt_eles_sz, 
               hipMemcpyDeviceToHost);
    hipMemcpy(args.centroid_elements, dev_centroid_elements, cent_eles_sz, 
               hipMemcpyDeviceToHost);
    hipMemcpy(args.labels, dev_labels, labels_sz, 
               hipMemcpyDeviceToHost);
//     hipEventRecord(end_stop);
//     hipEventSynchronize(end_stop);
//     hipEventElapsedTime(&end_diff, end_start, end_stop);
//     hipEventRecord(all_stop);
//     hipEventSynchronize(all_stop);
//     hipEventElapsedTime(&all_diff, all_start, all_stop);
    
    hipFree(dev_centroid_elements);
    hipFree(old_centroid_elements);
    hipFree(dev_point_elements);
    hipFree(dev_counts);
    
//     hipEventDestroy(all_start);
//     hipEventDestroy(all_stop);
//     hipEventDestroy(init_start);
//     hipEventDestroy(init_stop);
//     hipEventDestroy(end_start);
//     hipEventDestroy(end_stop);
    hipEventDestroy(iter_start);
    hipEventDestroy(iter_stop);
//     hipEventDestroy(tran1_start);
//     hipEventDestroy(tran1_stop);
//     hipEventDestroy(tran2_start);
//     hipEventDestroy(tran2_stop);
//     cout << "End-to-end: " << all_diff << " ms\n";
//     cout << "Initial transfer: " << init_diff << " ms\n";
//     cout << "Final transfer: " << end_diff << " ms\n";
//     cout << "Total tran1: " << tran1_diff << " ms\n";
//     cout << "Total tran2: " << tran2_diff << " ms\n";
//     cout << "Total iter: " << iter_diff << " ms\n";
}